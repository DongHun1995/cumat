#include "hip/hip_runtime.h"
#include <cstdio>
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

#define TS 32
#define WIDTH 4

static __global__ void matmul_kernel(float4 *A, float4 *B, float4 *C, int M, int N, int K)
{  
  int A_globalRow = blockDim.y * blockIdx.y + threadIdx.y;
  int B_globalCol = blockDim.x * blockIdx.x + threadIdx.x;
  int localRow = threadIdx.y;
  int localCol = threadIdx.x;

  __shared__ float4 Alocal[TS][TS/WIDTH];
  __shared__ float4 Blocal[TS][TS/WIDTH];

  float4 inter_val = { 0.0f, 0.0f, 0.0f, 0.0f};
  
  for (int bk =0; bk < K; bk += TS)
  {
    int A_globalCol = (bk/WIDTH) + localCol;
    int B_globalRow = bk + localRow;

    Alocal[localRow][localCol] = A[A_globalRow * (K / WIDTH) + A_globalCol];
    Blocal[localRow][localCol] = B[B_globalRow * (N / WIDTH) + B_globalCol];
    
    __syncthreads();

    float4 vecA, vecB;
    float valA;
    for (int k =0; k < TS/WIDTH; k++)
    {
      vecA = Alocal[localRow][k];
      for (int w =0; w < WIDTH; w++)
      {
        vecB = Blocal[WIDTH*k + w][localCol];

        switch(w)
        {
          case 0: valA = vecA.x; break;
          case 1: valA = vecA.y; break;
          case 2: valA = vecA.z; break;
          case 3: valA = vecA.w; break;
        }

        inter_val.x += vecB.x * valA;
        inter_val.y += vecB.y * valA;
        inter_val.z += vecB.z * valA;
        inter_val.w += vecB.w * valA;
      }
    }

    __syncthreads();
  }
  
    C[A_globalRow * (N / WIDTH) + B_globalCol] = inter_val;
  
}

#define NGPU 4
static size_t Mbegin[NGPU], Mend[NGPU];
static hipStream_t streams[NGPU];

static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];

static int mpi_rank, mpi_world_size;
int node_M;

#define SLICE 4
MPI_Request reqA[SLICE], reqB, req[50], gar[10];
int reqNum;

void matmul_initialize(int M, int N, int K) 
{
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
  node_M = M / SLICE / mpi_world_size;

  for (size_t i = 0; i < NGPU; i++)
  {
    Mbegin[i] = node_M / NGPU * i;
    Mend[i] = node_M / NGPU * (i + 1);
    if (i == NGPU - 1) Mend[i] = node_M;
  }

  for (size_t i = 0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  for (size_t i =0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMalloc(&A_gpu[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }

}

void matmul_slice(float *A, float *C, int M, int N, int K, int buf)
{
  if (mpi_rank != 0)
  {
    MPI_Wait(&reqA[buf], MPI_STATUS_IGNORE);
  }

  for (int i = 0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &A[Mbegin[i] * K], (Mend[i] - Mbegin[i]) * K * sizeof(float), hipMemcpyHostToDevice, streams[i]));    
  }

  for (int i = 0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    dim3 blockDim(TS / WIDTH, TS);
    dim3 gridDim((N + TS - 1) / TS, (Mend[i] - Mbegin[i] + TS - 1) / TS);
    matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(reinterpret_cast<float4*>(A_gpu[i]), reinterpret_cast<float4*>(B_gpu[i]), reinterpret_cast<float4*>(C_gpu[i]), Mend[i] - Mbegin[i], N, K);
    CHECK_CUDA(hipGetLastError());
  }

  for(size_t i =0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(&C[Mbegin[i] * N], C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float), hipMemcpyDeviceToHost, streams[i]));
  }

  for (size_t i =0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamSynchronize(streams[i]));
  }

  //MPI gather C
  if(mpi_rank == 0)
  {
    for (int i = 1; i < mpi_world_size; i++)
    {
      MPI_Irecv(&C[i * node_M * N], node_M * N, MPI_FLOAT, i, 0, MPI_COMM_WORLD, &req[reqNum++]);
    } 
  } else {
    MPI_Isend(C, node_M * N, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &gar[0]);
    MPI_Request_free(&gar[0]);
  }
}

void matmul(const float *A, const float *B, float *C, int M, int N, int K) 
{
  reqNum = 0;

  // SEND B
  if(mpi_rank == 0)
  {
    for (int i = 1; i < mpi_world_size; i++)
    {
      MPI_Isend(B, K * N, MPI_FLOAT, i, 0, MPI_COMM_WORLD, &gar[i - 1]);
      MPI_Request_free(&gar[i - 1]);

    }
  } else
  {
    MPI_Irecv((float *)B, K * N, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &reqB);
  }

  // SEND A
  for (int buf = 0; buf < SLICE; buf++)
  {
    int offset = buf * M / SLICE;

    if (mpi_rank == 0)
    {
      for (int i = 1; i < mpi_world_size; i++)
      {
        MPI_Isend(&A[offset * K + i * node_M * K], node_M * K, MPI_FLOAT, i, 0, MPI_COMM_WORLD, &gar[i-1]);
        MPI_Request_free(&gar[i-1]);

      }
    } else
    {
      MPI_Irecv((float *)&A[offset * K], node_M * K, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &reqA[buf]);
    }
  }

    // node B to gpu_B
  if (mpi_rank != 0)
  {
    MPI_Wait(&reqB, MPI_STATUS_IGNORE);
  }

  for (int i = 0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(B_gpu[i], B, K * N * sizeof(float), hipMemcpyHostToDevice, streams[i]));
  }

  for (int buf =0; buf < SLICE; buf++)
  {
    int offset = buf * M / SLICE;
    matmul_slice((float *)&A[offset * K], &C[offset * N], M / SLICE, N, K, buf);
  }

  MPI_Waitall(reqNum, req, MPI_STATUS_IGNORE);

}

void matmul_finalize() 
{
  for(size_t i = 0; i < NGPU; i++)
  {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
  }
}